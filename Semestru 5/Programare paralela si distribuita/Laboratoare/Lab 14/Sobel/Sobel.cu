#include "hip/hip_runtime.h"
//
// Sobel Edge Detection Filter using CUDA
//
#include "opencv2/imgproc/imgproc.hpp"
#include <opencv2/highgui.hpp>
#include <iostream>
#include <string>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

using namespace std;


//extern "C" bool sobelFilter_GPU_wrapper(const cv::Mat& input, cv::Mat& output);
extern "C" bool sobelFilter_CPU(const cv::Mat& input, cv::Mat& output);

// Run Sobel Edge Detect Filter on GPU
__global__ void sobelFilter(unsigned char *srcImage, unsigned char *dstImage, unsigned int width, unsigned int height)
{
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;

   float Kx[3][3] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
   float Ky[3][3] = {1, 2, 1, 0, 0, 0, -1, -2, -1};

   // only threads inside image will write results
   if((x>=FILTER_WIDTH/2) && (x<(width-FILTER_WIDTH/2)) && (y>=FILTER_HEIGHT/2) && (y<(height-FILTER_HEIGHT/2)))
   {
         // Gradient in x-direction 
         float Gx = 0;
         // Loop inside the filter to average pixel values
         for(int ky=-FILTER_HEIGHT/2; ky<=FILTER_HEIGHT/2; ky++) {
            for(int kx=-FILTER_WIDTH/2; kx<=FILTER_WIDTH/2; kx++) {
               float fl = srcImage[((y+ky)*width + (x+kx))];
               Gx += fl*Kx[ky+FILTER_HEIGHT/2][kx+FILTER_WIDTH/2];
            }
         }
         float Gx_abs = Gx < 0 ? -Gx : Gx;

         // Gradient in y-direction 
         float Gy = 0;
         // Loop inside the filter to average pixel values
         for(int ky=-FILTER_HEIGHT/2; ky<=FILTER_HEIGHT/2; ky++) {
            for(int kx=-FILTER_WIDTH/2; kx<=FILTER_WIDTH/2; kx++) {
               float fl = srcImage[((y+ky)*width + (x+kx))];
               Gy += fl*Ky[ky+FILTER_HEIGHT/2][kx+FILTER_WIDTH/2];
            }
         }
         float Gy_abs = Gy < 0 ? -Gy : Gy;

         dstImage[(y*width+x)] =  Gx_abs + Gy_abs;
   }
}


// The wrapper is use to call sobel edge detection filter 
extern "C" void sobelFilter_GPU_wrapper(const cv::Mat& input, cv::Mat& output)
{
        // Use cuda event to catch time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Calculate number of input & output bytes in each block
        const int inputSize = input.cols * input.rows;
        const int outputSize = output.cols * output.rows;
        unsigned char *d_input, *d_output;
        
        // Allocate device memory
        hipMalloc<unsigned char>(&d_input,inputSize);
        hipMalloc<unsigned char>(&d_output,outputSize);

        // Copy data from OpenCV input image to device memory
        hipMemcpy(d_input,input.ptr(),inputSize,hipMemcpyHostToDevice);

        // Specify block size
        const dim3 block(BLOCK_SIZE,BLOCK_SIZE);

        // Calculate grid size to cover the whole image
        const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y - 1)/block.y);

        // Start time
        hipEventRecord(start);

        // Run Sobel Edge Detection Filter kernel on CUDA 
        sobelFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows);

        // Stop time
        hipEventRecord(stop);

        //Copy data from device memory to output image
        hipMemcpy(output.ptr(),d_output,outputSize,hipMemcpyDeviceToHost);

        //Free the device memory
        hipFree(d_input);
        hipFree(d_output);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        
        // Calculate elapsed time in milisecond  
        hipEventElapsedTime(&milliseconds, start, stop);
        cout<< "\nProcessing time on GPU (ms): " << milliseconds << "\n";
}


// Program main
int main( int argc, char** argv ) {

   // name of image
   string image_name = "sample";

   // input & output file names
   string input_file =  image_name+".jpeg";
   string output_file_cpu = image_name+"_cpu.jpeg";
   string output_file_gpu = image_name+"_gpu.jpeg";

   // Read input image 
   cv::Mat srcImage = cv::imread(input_file ,CV_LOAD_IMAGE_UNCHANGED);
   if(srcImage.empty())
   {
      std::cout<<"Image Not Found: "<< input_file << std::endl;
      return -1;
   }
   cout <<"\ninput image size: "<<srcImage.cols<<" "<<srcImage.rows<<" "<<srcImage.channels()<<"\n";

   // convert RGB to gray scale
   cv::cvtColor(srcImage, srcImage, CV_BGR2GRAY);
  
   // Declare the output image  
   cv::Mat dstImage (srcImage.size(), srcImage.type());

   // run sobel edge detection filter on GPU  
   sobelFilter_GPU_wrapper(srcImage, dstImage);
   // normalization to 0-255
   dstImage.convertTo(dstImage, CV_32F, 1.0 / 255, 0);
   dstImage*=255;
   // Output image
   imwrite(output_file_gpu, dstImage);

   // run sobel edge detection filter on CPU  
   sobelFilter_CPU(srcImage, dstImage);
   // normalization to 0-255
   dstImage.convertTo(dstImage, CV_32F, 1.0 / 255, 0);
   dstImage*=255;
   // Output image
   imwrite(output_file_cpu, dstImage);
      
   return 0;
}




